#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <stdlib.h>
#include <unistd.h>

extern "C" __global__
void memcpy_kernel(unsigned char* __restrict__ output, const unsigned char* __restrict__ input){
    output += (blockIdx.x<<13)|(threadIdx.x<<2);
    input  += (blockIdx.x<<13)|(threadIdx.x<<2);
    *((float* )&output[0])       = *((float* )&input[0]);
    *((float* )&output[0x400])   = *((float* )&input[0x400]);
    *((float* )&output[0x800])   = *((float* )&input[0x800]);
    *((float* )&output[0xc00])   = *((float* )&input[0xc00]);
    *((float* )&output[0x1000])  = *((float* )&input[0x1000]);
    *((float* )&output[0x1400])  = *((float* )&input[0x1400]);
    *((float* )&output[0x1800])  = *((float* )&input[0x1800]);
    *((float* )&output[0x1c00])  = *((float* )&input[0x1c00]);
}

#define CALL(cmd) \
do {\
    hipError_t cuda_error  = cmd;\
    if (cuda_error != hipSuccess) { \
        std::cout<<"'"<<hipGetErrorString(cuda_error)<<"'("<<cuda_error<<")"<<" at "<<__FILE__<<":"<<__LINE__<<std::endl;\
        exit(EXIT_FAILURE);\
    }\
} while(0)

#define WARMUP 2
#define LOOP 10

static inline void b2s(size_t bytes, char * str){
	if(bytes<1024){
		sprintf(str, "%luB", bytes);
	}else if(bytes<(1024*1024)){
		double b= (double)bytes/1024.0;
		sprintf(str, "%.2fKB", b);
	}else if(bytes<(1024*1024*1024)){
		double b= (double)bytes/(1024.0*1024);
		sprintf(str, "%.2fMB", b);
	}else{
		double b= (double)bytes/(1024.0*1024*1024);
		sprintf(str, "%.2fGB", b);
	}
}

int main() {
	hipSetDevice(0);
    unsigned char *A, *B;
    const int total_float =64*3*224*224;
	static float h_A[total_float], h_B[total_float];
	for (int i = 0; i < total_float; ++i)
		h_A[i] = i % 71;

    CALL(hipMalloc(&A, total_float * sizeof(float)));
    CALL(hipMalloc(&B, total_float * sizeof(float)));
    CALL(hipMemcpy(A, h_A, total_float * sizeof(float), hipMemcpyHostToDevice));

    // benchmark kernel
    int bx = 256;
    int gx = (total_float+255)>>11;
    assert(total_float/bx);

    hipEvent_t start_ev, stop_ev;
    CALL(hipEventCreate(&start_ev));
    CALL(hipEventCreate(&stop_ev));

    for(int i=0;i<WARMUP;i++)
        memcpy_kernel<<<gx, bx>>>(B, A);

    CALL(hipEventRecord( start_ev, 0));
    for(int i=0;i<LOOP;i++)
        memcpy_kernel<<<gx, bx>>>(B, A);
    CALL(hipEventRecord( stop_ev, 0 ));
    CALL(hipEventSynchronize(stop_ev));

    float ms;
    CALL(hipEventElapsedTime(&ms,start_ev, stop_ev));
    ms/=LOOP;

    sleep(1);

    // benchmark memcpy api
    for(int i=0;i<WARMUP;i++)
        CALL(hipMemcpy(B, A, total_float * sizeof(float), hipMemcpyDeviceToDevice));
    CALL(hipEventRecord( start_ev, 0));
    for(int i=0;i<LOOP;i++)
        CALL(hipMemcpy(B, A, total_float * sizeof(float), hipMemcpyDeviceToDevice));
    CALL(hipEventRecord( stop_ev, 0 ));
    CALL(hipEventSynchronize(stop_ev));

    float ms_api;
    CALL(hipEventElapsedTime(&ms_api,start_ev, stop_ev));
    ms_api/=LOOP;

    char str[64];
    b2s(total_float*sizeof(float), str);
    printf("%s, gflops_kernel:%.3f, gflops_api:%.3f\n", str, ((double)total_float*sizeof(float)*2)/((double)ms/1000)/1000000000.0,
    ((double)total_float*sizeof(float)*2)/((double)ms_api/1000)/1000000000.0 );
}
